#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
using namespace std;

#define BLOCK_SIZE 32
// ���, ������� ����� ����� �������� ������
#define BASE_TYPE float
#define THREADS_PER_BLOCK 128
// ������ 
#define N BLOCK_SIZE * THREADS_PER_BLOCK


__constant__ float constDataA[N];
__constant__ float constDataB[N];



__global__ void scalMult2(BASE_TYPE *C)
{
	__shared__ BASE_TYPE ash[BLOCK_SIZE];
	__shared__ BASE_TYPE bsh[BLOCK_SIZE];
	ash[threadIdx.x] = constDataA[blockIdx.x * blockDim.x + threadIdx.x] * constDataB[blockIdx.x * blockDim.x + threadIdx.x];
	__syncthreads();
	if (threadIdx.x == 0)
	{
		BASE_TYPE sum = 0.0;
		for (int j = 0; j < blockDim.x; j++)
		{
			sum += ash[j];
		}
		C[blockIdx.x] = sum;
	}
}

int main()
{
	BASE_TYPE *h_a;
	h_a = (BASE_TYPE*)malloc(N * sizeof(BASE_TYPE));
	BASE_TYPE *h_b;
	h_b = (BASE_TYPE *)malloc(N * sizeof(BASE_TYPE));
	BASE_TYPE h_c = 0;
	for (int i = 0; i< N; i++)
	{
		h_a[i] = 1;// rand() % 10 + 1;
				   //printf("h_a[%f] = %f\n", i, h_a[i]);
		h_b[i] = 1;// rand() % 10 + 1;
				   //printf("h_b[%f] = %f\n", i, h_b[i]);
	}
	printf("scalar on host:\n");
	for (int i = 0; i != N; i++)
	{
		BASE_TYPE temp = h_a[i] * h_b[i];
		h_c += temp;
	}
	printf("host result = %f\n", h_c);

	h_c = 0;

	BASE_TYPE * d_c;
	hipMalloc((void**)&d_c, N * sizeof(BASE_TYPE));
	hipMemcpy(d_c, &h_c, sizeof(BASE_TYPE), hipMemcpyHostToDevice);
	// ����������� ������ � ������������ ���������� �
	// ����������� ������
	hipMemcpyToSymbol(HIP_SYMBOL(constDataA), h_a, N * sizeof(float), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(constDataB), h_b, N * sizeof(float), 0, hipMemcpyHostToDevice);
	hipEvent_t start, stop;
	float elapsedTime;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	scalMult2 << < BLOCK_SIZE, THREADS_PER_BLOCK >> > (d_c);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Time spent executing by the GPU: %.2f millseconds\n", elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	BASE_TYPE *h_cc = (BASE_TYPE*)malloc(N * sizeof(BASE_TYPE));
	hipMemcpy(h_cc, d_c, N * sizeof(BASE_TYPE), hipMemcpyDeviceToHost);


	for (int i = 1; i != N; i++)
	{
		h_cc[0] += h_cc[i];
	}
	printf("h_cc[%i] = %f\n", 0, h_cc[0]);

	hipDeviceSynchronize();
	getchar();
	return 0;
}
