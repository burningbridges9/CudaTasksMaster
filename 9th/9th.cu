#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
#include ""
// ����������� ���������� cuBLAS
#include <hipblas.h>
#include <cublasXt.h>


#define IDX2C(i,j,ld) (((i)*(ld))+(j))

__global__ void show(float **A, int cols, int rows)
{
	printf("Matrix on GPU:\n");
	for (int i = 0; i != rows; i++)
	{
		for (int j = 0; j != cols; j++)
		{
			printf("%.1f  ", A[i][j]);
		}
		printf("\n");
	}
}

void CublasExample()
{
	//int batch_size = 1;
	hipError_t cudaStat;
	hipblasStatus_t stat;
	hipblasHandle_t handle;

	const int N = 6;
	float *dev_A, *dev_b, **dev_Aarray;
	float *x, *A, *b, **Aarray;
	/*int * info_array;
	int * dev_info_array;*/
	x = (float *)malloc(N * sizeof(*x));
	if (!x) {
		printf("host memory allocation failed");
		//return EXIT_FAILURE;
	}
	b = (float *)malloc(N * sizeof(*b));
	if (!b) {
		printf("host memory allocation failed");
		//return EXIT_FAILURE;
	}
	A = (float *)malloc(N * N * sizeof(*A));
	if (!A) {
		printf("host memory allocation failed");
		//return EXIT_FAILURE;
	}
	/*info_array = (int *)malloc(batch_size * sizeof(*info_array));
	Aarray = (float **)malloc(sizeof(float*));
	Aarray[0] = (float *)malloc(N * N * sizeof(*A));*/
	// ������������� ������� � ������� ������ �����
	int ind = 11;
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
			if (i >= j)
				A[IDX2C(i, j, N)] = (float)ind++;
			else A[IDX2C(i, j, N)] = 0.0f;
			b[i] = 1.0f;
	}
#pragma region  HomeFirst

	//Aarray[0] = A;
	//printf("Generated matrix A:\n");
	//for (int k = 0; k < 1; k++)
	//{
	//	for (int i = 0; i < N; i++)
	//	{
	//		for (int j = 0; j < N; j++)
	//			printf("A[%d][%d] = %f  ", i, j, Aarray[k][IDX2C(i, j, N)]);
	//		printf("\n");
	//	}
	//}


	//hipMalloc((void **)&dev_Aarray, 1 * sizeof(float*)); // �������� ������ �� ������ �� ���������� �� float
	//float **h_tempA = (float **)malloc(1 * sizeof(float*));
	//for (int i = 0; i<1; i++) {
	//	hipMalloc((void**)&h_tempA[i], N * N * sizeof(float));
	//}
	//for (int i = 0; i<1; i++) {
	//	hipMemcpy(h_tempA[i], Aarray[i], N * sizeof(float), hipMemcpyHostToDevice);
	//}
	//hipMemcpy(dev_Aarray, h_tempA, 1 * sizeof(float*), hipMemcpyHostToDevice);
	/*printf("A matrix:\n");
	show << <1, 1 >> >(dev_Aarray, N*N, 1);

	printf("Generated matrix Aarray:\n");
	for (int i = 0; i < N; i++)
	{
	for (int j = 0; j < N; j++)
	printf("A[%d][%d] = %f  ", i, j, A[IDX2C(i, j, N)]);
	printf("\n");
	}*/

#pragma endregion
	
	for (int j = 0; j < N; j++)
		printf("B[%d] = %f  ", j, b[j]);
	// �������� ������ �� GPU ���������������� �������
	// ��� ������ ����������
	cudaStat =hipMalloc((void**)&dev_b, N * sizeof(*x));
	if (cudaStat != hipSuccess) {
		printf("device memory allocation failed");
		//return EXIT_FAILURE;
	}
	cudaStat =hipMalloc((void**)&dev_A, N * N * sizeof(float));
	if (cudaStat != hipSuccess) {
		printf("device memory allocation failed");
		//return EXIT_FAILURE;
	}
	//hipMalloc((void**)&dev_info_array, batch_size* sizeof(int));
	// �������������� �������� cuBLAS
	stat = hipblasCreate(&handle);
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		printf("CUBLAS initialization failed\n");
		//return EXIT_FAILURE;
	}
	// �������� ������ � ������� �� CPU � GPU
	stat =  hipblasSetVector(N, sizeof(*b), b, 1, dev_b, 1);
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		printf("CUBLAS set vector failed\n");
		//return EXIT_FAILURE;
	}
#pragma region HomeFirst
	//hipblasSetVector(batch_size, sizeof(*info_array), info_array, 1, dev_info_array, 1);
	/*hipblasSetMatrix(N, N, sizeof(*A), A, N, dev_A, N);

	hipblasSgetrfBatched(handle,
	1,
	dev_Aarray,
	6,
	NULL,
	dev_info_array,
	1);*/
#pragma endregion

	stat = hipblasSetMatrix(N, N, sizeof(*A), A, N, dev_A, N);
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		printf("CUBLAS set matrix failed\n");
		//return EXIT_FAILURE;
	}
	// ������ ������ ���������� �������
	stat = hipblasStrsv(handle, HIPBLAS_FILL_MODE_LOWER,
		HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, N, dev_A, N,
		dev_b, 1); 
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		printf("CUBLAS solve failed\n");
		//return EXIT_FAILURE;
	}
	// �������� ��������� �� GPU � CPU
	stat = hipblasGetVector(N, sizeof(*x), dev_b, 1, x, 1);
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		printf("CUBLAS get vector failed\n");
		//return EXIT_FAILURE;
	}
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
			printf("%3.0f ", A[IDX2C(i, j, N)]);
		printf(" = %f %4.6f\n", b[i], x[i]);
	}
	// ����������� ������ � GPU
	hipFree(dev_b);
	hipFree(dev_A);
	// ���������� ������� cuBLAS
	hipblasDestroy(handle);
	// ����������� ������ � CPU
	free(x);
	free(b);
	free(A);
}

void MultiCublas(int N)
{
	hipError_t cudaStat;
	hipblasStatus_t stat;
	hipblasHandle_t handle;

	float *dev_A, *dev_B;
	float *A, *B;
	B = (float *)malloc(N * N * sizeof(*B));
	if (!B) {
		printf("host memory allocation failed");
		//return EXIT_FAILURE;
	}
	A = (float *)malloc(N * N * sizeof(*A));
	if (!A) {
		printf("host memory allocation failed");
		//return EXIT_FAILURE;
	}
	// ������������� ������� � ������� ������ �����
	int ind = 11;
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			A[IDX2C(i, j, N)] = (float)ind++;
			B[IDX2C(i, j, N)] = (float)ind++;
		}
	}
	//printf("A gen:\n");
	//for (int i = 0; i < N; i++)
	//{
	//	for (int j = 0; j < N; j++)
	//		printf("%0.0f ", A[IDX2C(i, j, N)]);
	//	printf("\n");
	//}
	//printf("B gen:\n");
	//for (int i = 0; i < N; i++)
	//{
	//	for (int j = 0; j < N; j++)
	//		printf("%0.0f ", B[IDX2C(i, j, N)]);
	//	printf("\n");
	//}

	// �������� ������ �� GPU ���������������� �������
	// ��� ������ ����������
	cudaStat = hipMalloc((void**)&dev_B, N * N*sizeof(float));
	if (cudaStat != hipSuccess) {
		printf("device memory allocation failed");
		//return EXIT_FAILURE;
	}
	cudaStat = hipMalloc((void**)&dev_A, N * N * sizeof(float));
	if (cudaStat != hipSuccess) {
		printf("device memory allocation failed");
		//return EXIT_FAILURE;
	}

	// �������������� �������� cuBLAS
	stat = hipblasCreate(&handle);
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		printf("CUBLAS initialization failed\n");
		//return EXIT_FAILURE;
	}
	// �������� ������ � ������� �� CPU � GPU
	stat = hipblasSetMatrix(N, N, sizeof(*B), B, N, dev_B, N);
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		printf("CUBLAS set matrix failed\n");
		//return EXIT_FAILURE;
	}

	stat = hipblasSetMatrix(N, N, sizeof(*A), A, N, dev_A, N);
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		printf("CUBLAS set matrix failed\n");
		//return EXIT_FAILURE;
	}

	// �������������� �������
	hipEvent_t start, stop;
	float elapsedTime;
	// ������� �������
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// A*B = A
	float alpha = 1;
	float beta = 0;
	// ������ �������
	hipEventRecord(start, 0);
	
	hipblasSgemm(handle,
		HIPBLAS_OP_N, HIPBLAS_OP_N,
		N, N, N,
		&alpha,
		dev_A, N,
		dev_B, N,
		&beta,
		dev_A, N);
	hipEventRecord(stop, 0);
	// �������� ���������� ������ ����
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	// ����� ����������
	printf("-----------------------------\n");
	printf("CUBLAS; Time spent executing by the GPU: %.2f millseconds\n", elapsedTime);
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		printf("CUBLAS solve failed\n");
		//return EXIT_FAILURE;
	}
	// �������� ��������� �� GPU � CPU
	
	stat = hipblasGetMatrix(N, N, sizeof(*A),
		dev_A, N, A, N);
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		printf("CUBLAS get vector failed\n");
		//return EXIT_FAILURE;
	}
	//printf("Res:\n");
	//for (int i = 0; i < N; i++)
	//{
	//	for (int j = 0; j < N; j++)
	//		printf("%0.0f ", A[IDX2C(i, j, N)]);
	//	printf("\n");
	//}
	// ����������� ������ � GPU
	hipFree(dev_B);
	hipFree(dev_A);
	// ���������� ������� cuBLAS
	hipblasDestroy(handle);
	// ����������� ������ � CPU
	free(B);
	free(A);
	// ����������� �������
	hipEventDestroy(start);
	hipEventDestroy(stop);

}

void MultiCublasXt(int N)
{
#pragma region Init
	hipError_t cudaStat;
	hipblasStatus_t stat;
	cublasXtHandle_t handle;
	int devices[1] = { 0 };
	// �������������� �������� CUBLAS-XT
	stat = cublasXtCreate(&handle);
	if (stat != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "!!!! CUBLASXT initialization error\n");
		//return EXIT_FAILURE;
	}
	/* �������� ���������� ��� ������� ������� CUBLAS-XT
	����� �������� ����� �������������� �������� */
	stat = cublasXtDeviceSelect(handle, 1, devices);
	if (stat != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "!!!! CUBLASXT device selection	error\n");
		//return EXIT_FAILURE;
	}
	/* ������������� ������ ������ (blockDim x blockDim) ��
	������� ����� ����������� ������� ��� ������������� �����
	������������ */
	stat = cublasXtSetBlockDim(handle, 64);
	if (stat != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "!!!! CUBLASXT set block dimension error\n");
		//return EXIT_FAILURE;
	}
	float *dev_A, *dev_B;
	float *A, *B, *C;
	B = (float *)malloc(N * N * sizeof(*B));
	if (!B) {
		printf("host memory allocation failed");
		//return EXIT_FAILURE;
	}
	A = (float *)malloc(N * N * sizeof(*A));
	if (!A) {
		printf("host memory allocation failed");
		//return EXIT_FAILURE;
	}
	C = (float *)malloc(N * N * sizeof(*C));
	if (!C) {
		printf("host memory allocation failed");
		//return EXIT_FAILURE;
	}
	// ������������� ������� � ������� ������ �����
	int ind = 11;
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			A[IDX2C(i, j, N)] = (float)ind++;
			B[IDX2C(i, j, N)] = (float)ind++;
		}
	}
	/*printf("A gen:\n");
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
			printf("%0.0f ", A[IDX2C(i, j, N)]);
		printf("\n");
	}
	printf("B gen:\n");
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
			printf("%0.0f ", B[IDX2C(i, j, N)]);
		printf("\n");
	}*/
#pragma endregion	
	// �������������� �������
	hipEvent_t start, stop;
	float elapsedTime;
	// ������� �������
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// A*B = A
	float alpha = 1;
	float beta = 0;
	// ������ �������
	hipEventRecord(start, 0);

	stat = cublasXtSgemm(handle,
		HIPBLAS_OP_N, HIPBLAS_OP_N,
		N, N, N,
		&alpha,
		A, N,
		B, N,
		&beta,
		C, N);
	hipEventRecord(stop, 0);
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		printf("CUBLAS initialization failed\n");
		//return EXIT_FAILURE;
	}
	// �������� ���������� ������ ����
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	// ����� ����������
	printf("-----------------------------\n");
	printf("CUBLASXT; Time spent executing by the GPU: %.2f millseconds\n", elapsedTime);
	printf("-----------------------------\n");
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		printf("CUBLAS solve failed\n");
		//return EXIT_FAILURE;
	}
	//printf("Res:\n");
	//for (int i = 0; i < N; i++)
	//{
	//	for (int j = 0; j < N; j++)
	//		printf("%0.0f ", C[IDX2C(i, j, N)]);
	//	printf("\n");
	//}
	cublasXtDestroy(handle);
	free(B);
	free(A);
	hipEventDestroy(start);
	hipEventDestroy(stop);

}


__global__ void matrixMultiplicationKernel(float* A, float* B, float* C, int N) 
{
	int ROW = blockIdx.y*blockDim.y + threadIdx.y;
	int COL = blockIdx.x*blockDim.x + threadIdx.x;
	float tmpSum = 0;
	
	if (ROW < N && COL < N) {
		// each thread computes one element of the block sub-matrix
		for (int i = 0; i < N; i++) {
			tmpSum += A[ROW * N + i] * B[i * N + COL];
		}

		C[ROW * N + COL] = tmpSum;
	}
}

void MultiSimple(int N)
{
	hipEvent_t start, stop;
	float elapsedTime;
	// ������� �������
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int threads_num = 10;
	int blocks_num = N / threads_num;
	dim3 threadsPerBlock = dim3(threads_num, threads_num);
	dim3 blocksPerGrid = dim3(N/blocks_num, N/blocks_num);

	float *h_A = (float *)malloc(N*N * sizeof(float));
	float *h_B = (float *)malloc(N*N * sizeof(float));
	float *h_C = (float *)malloc(N*N * sizeof(float));
	int ind = 11;
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			h_A[IDX2C(i, j, N)] = (float)ind++;
			h_B[IDX2C(i, j, N)] = (float)ind++;
			h_C[IDX2C(i, j, N)] = 0;
		}
	}
	/*printf("A matrix:\n");
	for (int i = 0; i != N; i++)
	{
		for (int j = 0; j != N; j++)
		{
			printf("%.1f ", h_A[i*N + j]);
			printf("%.1f ", h_B[i*N + j]);
		}
		printf("\n");
	}*/

	float *d_A = NULL;
	hipMalloc((void **)&d_A, N*N * sizeof(float));
	float * d_B = NULL;
	hipMalloc((void **)&d_B, N*N * sizeof(float));
	float * d_C = NULL;
	hipMalloc((void **)&d_C, N*N * sizeof(float));

	hipMemcpy(d_A, h_A, N*N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, N*N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_C, h_C, N*N * sizeof(float), hipMemcpyHostToDevice);

	hipEventRecord(start, 0);
	matrixMultiplicationKernel << <blocksPerGrid, threadsPerBlock >> > (d_A, d_B, d_C, N);
	hipEventRecord(stop, 0);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	hipMemcpy(h_C, d_C, N*N * sizeof(float), hipMemcpyDeviceToHost);
	/*for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			printf("%f ", h_C[IDX2C(i, j, N)]);
		}
		printf("\n");
	}*/
	printf("Simple; Time spent executing by the GPU: %.2f millseconds\n", elapsedTime);
	printf("-----------------------------\n");
	free(h_B);
	free(h_A);
	free(h_C);
	hipEventDestroy(start);
	hipEventDestroy(stop);
}

void ThirdHome()
{
	for (int i = 10; i <= 100; i += 10)
	{
		printf("N = %d\n", i);
		MultiCublas(i);
		MultiCublasXt(i);
		MultiSimple(i);
	}
}

#define N (2)
void FirstLab()
{
#pragma region Init CuBlasXt
	hipblasStatus_t status;
	cublasXtHandle_t handle;
	int devices[1] = { 0 };
	// �������������� �������� CUBLAS-XT
	status = cublasXtCreate(&handle);
	if (status != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "!!!! CUBLASXT initialization error\n");
		//return EXIT_FAILURE;
	}
	/* �������� ���������� ��� ������� ������� CUBLAS-XT
	����� �������� ����� �������������� �������� */
	status = cublasXtDeviceSelect(handle, 1, devices);
	if (status != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "!!!! CUBLASXT device selection	error\n");
		//return EXIT_FAILURE;
	}
	/* ������������� ������ ������ (blockDim x blockDim) ��
	������� ����� ����������� ������� ��� ������������� �����
	������������ */
	status = cublasXtSetBlockDim(handle, 64);
	if (status != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "!!!! CUBLASXT set block dimension error\n");
		//return EXIT_FAILURE;
	}
#pragma endregion


	float *h_A;
	float *h_B;
	float *h_I;
	float *h_ABSum;
	float *h_C;
	// �������
	float alpha = 1.0f;
	float beta = 1.0f;
	// ������ �������
	int n2 = N * N;
	
	// �������� ������ ��� ������ � ��������� ������
	h_A = (float *)malloc(n2 * sizeof(h_A[0]));
	h_B = (float *)malloc(n2 * sizeof(h_B[0]));
	h_ABSum = (float *)malloc(n2 * sizeof(h_ABSum[0]));
	h_I = (float *)malloc(n2 * sizeof(h_I[0]));
	h_C = (float *)malloc(n2 * sizeof(h_C[0]));
	// ��������� ������� ��������� �������
	srand((unsigned int)time(NULL));
	for (int i = 0; i < n2; i++)
	{
		h_A[i] = rand() / (float)RAND_MAX;
		h_B[i] = rand() / (float)RAND_MAX;
		h_ABSum[i] = h_B[i];
	}
	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
			h_I[i*N+j] = i == j ? 1 : 0;

#pragma region Print
	printf("initialized A:\n");
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
			printf("A[%d][%d] = %f  ", i, j, h_A[i*N + j]);
		printf("\n");
	}
	printf("initialized B:\n");
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
			printf("B[%d][%d] = %f  ", i, j, h_B[i*N + j]);
		printf("\n");
	}
	printf("initialized I:\n");
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
			printf("I[%d][%d] = %f  ", i, j, h_I[i*N + j]);
		printf("\n");
	}
#pragma endregion


	// ��������� �������� ������������ ������
	printf("1*A*I + 1*B :\n");
	status = cublasXtSgemm(handle, HIPBLAS_OP_N,
		HIPBLAS_OP_N, N, N, N, &alpha, h_A, N, h_I, N, &beta,
		h_ABSum, N);
	if (status != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "!!!! kernel execution error.\n");
		//return EXIT_FAILURE;
	}
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
			printf("ABSum[%d][%d] = %f  ", i, j, h_ABSum[i*N + j]);
		printf("\n");
	}
	beta = 0.0f;
	// ��������� �������� ������������ ������
	printf("1*A*B + 0*C :\n");
	status = cublasXtSgemm(handle, HIPBLAS_OP_N,
		HIPBLAS_OP_N, N, N, N, &alpha, h_A, N, h_B, N, &beta,
		h_C, N);
	if (status != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "!!!! kernel execution error.\n");
		//return EXIT_FAILURE;
	}
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
			printf("C[%d][%d] = %f  ", i, j, h_C[i*N + j]);
		printf("\n");
	}
	// ��������� �������� ������������ ������
	printf("1*(A+B) * (A*B) + 0*C :\n");
	status = cublasXtSgemm(handle, HIPBLAS_OP_N,
		HIPBLAS_OP_N, N, N, N, &alpha, h_ABSum, N, h_C, N, &beta,
		h_C, N);
	if (status != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "!!!! kernel execution error.\n");
		//return EXIT_FAILURE;
	}
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
			printf("C[%d][%d] = %f  ", i, j, h_C[i*N + j]);
		printf("\n");
	}
	
	free(h_A);
	free(h_B);
	free(h_C);
	free(h_ABSum);
}


void SecondHome()
{
#pragma region Init CuBlasXt
	hipblasStatus_t status;
	cublasXtHandle_t handle;
	int devices[1] = { 0 };
	// �������������� �������� CUBLAS-XT
	status = cublasXtCreate(&handle);
	if (status != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "!!!! CUBLASXT initialization error\n");
		//return EXIT_FAILURE;
	}
	/* �������� ���������� ��� ������� ������� CUBLAS-XT
	����� �������� ����� �������������� �������� */
	status = cublasXtDeviceSelect(handle, 1, devices);
	if (status != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "!!!! CUBLASXT device selection	error\n");
		//return EXIT_FAILURE;
	}
	/* ������������� ������ ������ (blockDim x blockDim) ��
	������� ����� ����������� ������� ��� ������������� �����
	������������ */
	status = cublasXtSetBlockDim(handle, 64);
	if (status != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "!!!! CUBLASXT set block dimension error\n");
		//return EXIT_FAILURE;
	}
#pragma endregion


	float *h_A;
	float *h_B;
	float *h_I;
	float *h_ACSum;
	float *h_C;
	// �������
	float alpha = 1.0f;
	float beta = 1.0f;
	// ������ �������
	int n2 = N * N;

	// �������� ������ ��� ������ � ��������� ������
	h_A = (float *)malloc(n2 * sizeof(h_A[0]));
	h_B = (float *)malloc(n2 * sizeof(h_B[0]));
	h_ACSum = (float *)malloc(n2 * sizeof(h_ACSum[0]));
	h_I = (float *)malloc(n2 * sizeof(h_I[0]));
	h_C = (float *)malloc(n2 * sizeof(h_C[0]));
	// ��������� ������� ��������� �������
	srand((unsigned int)time(NULL));

	h_A[0] = 1; h_A[1] = 1; h_A[2] = 0; h_A[3] = 1;
	h_C[0] = 1; h_C[1] = -1; h_C[2] = 0; h_C[3] = 1;
	h_B[0] = 12; h_B[1] = 10; h_B[2] = 6; h_B[3] = 8;

	for (int i = 0; i < n2; i++)
	{
		h_ACSum[i] = h_C[i];
	}
	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
			h_I[i*N + j] = i == j ? 1 : 0;

#pragma region Print
	printf("initialized A:\n");
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
			printf("A[%d][%d] = %f  ", i, j, h_A[i*N + j]);
		printf("\n");
	}
	printf("initialized B:\n");
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
			printf("B[%d][%d] = %f  ", i, j, h_B[i*N + j]);
		printf("\n");
	}
	printf("initialized C:\n");
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
			printf("C[%d][%d] = %f  ", i, j, h_C[i*N + j]);
		printf("\n");
	}
	printf("initialized I:\n");
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
			printf("I[%d][%d] = %f  ", i, j, h_I[i*N + j]);
		printf("\n");
	}
#pragma endregion


	// ��������� �������� ������������ ������
	printf("1*A*I + 1*C :\n");
	status = cublasXtSgemm(handle, HIPBLAS_OP_N,
		HIPBLAS_OP_N, N, N, N, &alpha, h_A, N, h_I, N, &beta,
		h_ACSum, N);
	if (status != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "!!!! kernel execution error.\n");
		//return EXIT_FAILURE;
	}
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
			printf("ACSum[%d][%d] = %f  ", i, j, h_ACSum[i*N + j]);
		printf("\n");
	}
	beta = 0.0f;
	// ��������� �������� ������������ ������
	printf("1*(A*C)*C + 0*C :\n");
	status = cublasXtSgemm(handle, HIPBLAS_OP_N,
		HIPBLAS_OP_N, N, N, N, &alpha, h_ACSum, N, h_C, N, &beta,
		h_C, N);
	if (status != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "!!!! kernel execution error.\n");
		//return EXIT_FAILURE;
	}
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
			printf("C[%d][%d] = %f  ", i, j, h_C[i*N + j]);
		printf("\n");
	}
	// ��������� ������ ������� �
	printf("1*(A+B) * (A*B) + 0*C :\n");
	cublasXtStrsm(handle,
		HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER,
		HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,
		N, N,
		&alpha,
		h_C, N,
		h_B, N);

	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
			printf("X[%d][%d] = %f  ", i, j, h_B[i*N + j]);
		printf("\n");
	}

	if (status != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "!!!! kernel execution error.\n");
		//return EXIT_FAILURE;
	}

	free(h_A);
	free(h_B);
	free(h_C);
	free(h_ACSum);
}


int main()
{	
	//FirstLab();
	//SecondHome();
	
	//CublasExample();
	ThirdHome();
	getchar();
	return EXIT_SUCCESS;
}