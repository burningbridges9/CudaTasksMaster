#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
using namespace std;
#define N 1024
#define THREADS_PER_BLOCK 256
#define BLOCK_NUM  16

__global__ void Eval(int *nCirc, float * x, float *y)
{
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	if (i < N)
	{
		if (x[i] * x[i] + y[i] * y[i] <= 1)
		{
			atomicAdd(nCirc, 1);
		}
	}
}

__global__ void Evalx(float * x, unsigned seed)
{
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	hiprandState_t t;
	hiprand_init(seed, i, 0, &t);
	if (i < N)
	{
		x[i] = hiprand_uniform(&t);
		//printf("x[%i]=%f\n", i, x[i]);
	}
}

__global__ void Evaly(float * y, unsigned seed)
{
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	printf("blockDim.x=%i\n", blockDim.x);
	printf("blockIdx.x=%i\n", blockIdx.x);
	hiprandState_t m;
	hiprand_init(seed, i, 0, &m);
	if (i < N)
	{
		y[i] = hiprand_uniform(&m);
		//printf("y[%i]=%f\n", i, y[i]);
	}
}

void PiCalculation()
{
	int sizeI = sizeof(int);
	int sizeF = sizeof(float);
	int nCirc = 0;
	float x[N];
	float y[N];
	int *devNcirc;
	float *devx, *devy;
	hipMalloc((void**)&devx, N*sizeF);
	hipMalloc((void**)&devy, N*sizeF);
	hipMalloc((void**)&devNcirc, sizeI);
	hipMemcpy(devNcirc, &nCirc, sizeI, hipMemcpyHostToDevice);
	Evalx << <BLOCK_NUM, THREADS_PER_BLOCK >> > (devx, 0);
	Evaly << <BLOCK_NUM, THREADS_PER_BLOCK >> > (devy, time(NULL));
	hipMemcpy(&x, devx, N*sizeF, hipMemcpyDeviceToHost);
	hipMemcpy(&y, devy, N*sizeF, hipMemcpyDeviceToHost);
	/*for (int i = 0; i != N; i++)
	{
	printf("x[%i]=%f\n", i, x[i]);
	printf("y[%i]=%f\n", i, y[i]);
	}*/
	Eval << <N / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> > (devNcirc, devx, devy);
	hipMemcpy(&nCirc, devNcirc, sizeI, hipMemcpyDeviceToHost);
	printf("pi = %f\n", (nCirc*4.0) / N);

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) printf("%s ",
		hipGetErrorString(err));

	hipFree(devNcirc);
	hipFree(devx);
	hipFree(devy);
}

int main()
{
	PiCalculation();
	hipDeviceSynchronize();
	getchar();
}
