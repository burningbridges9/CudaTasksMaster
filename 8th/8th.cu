#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
using namespace std;

#define BLOCK_SIZE 32
// ���, ������� ����� ����� �������� ������
#define BASE_TYPE float
#define THREADS_PER_BLOCK 128
// ������ 
#define N BLOCK_SIZE * THREADS_PER_BLOCK


__constant__ float constDataA[N];
__constant__ float constDataB[N];



__global__ void ScalMult(BASE_TYPE *C)
{
	__shared__ BASE_TYPE ash[THREADS_PER_BLOCK];
	ash[threadIdx.x] = constDataA[blockIdx.x * blockDim.x + threadIdx.x] * constDataB[blockIdx.x * blockDim.x + threadIdx.x];
	__syncthreads();
	if (threadIdx.x == 0)
	{
		BASE_TYPE sum = 0.0;
		for (int j = 0; j < THREADS_PER_BLOCK; j++)
		{
			sum += ash[j];
		}
		atomicAdd(C, sum);
	}
}

void FirstLab()
{
	BASE_TYPE *h_a;
	h_a = (BASE_TYPE*)malloc(N * sizeof(BASE_TYPE));
	BASE_TYPE *h_b;
	h_b = (BASE_TYPE *)malloc(N * sizeof(BASE_TYPE));
	BASE_TYPE h_c = 0;
	for (int i = 0; i< N; i++)
	{
		h_a[i] = rand() / (BASE_TYPE)RAND_MAX;// rand() % 10 + 1;
				   //printf("h_a[%f] = %f\n", i, h_a[i]);
		h_b[i] = rand() / (BASE_TYPE)RAND_MAX;// rand() % 10 + 1;
				   //printf("h_b[%f] = %f\n", i, h_b[i]);
	}
	printf("scalar on host:\n");
	for (int i = 0; i != N; i++)
	{
		BASE_TYPE temp = h_a[i] * h_b[i];
		h_c += temp;
	}
	printf("host result = %f\n", h_c);
	h_c = 0;
	BASE_TYPE * d_c;
	hipMalloc((void**)&d_c, N * sizeof(BASE_TYPE));
	hipMemcpy(d_c, &h_c, sizeof(BASE_TYPE), hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(constDataA), h_a, N * sizeof(float), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(constDataB), h_b, N * sizeof(float), 0, hipMemcpyHostToDevice);
	hipEvent_t start, stop;
	float elapsedTime;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	ScalMult << < BLOCK_SIZE, THREADS_PER_BLOCK >> > (d_c);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Time spent executing by the GPU: %.2f millseconds\n", elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipMemcpy(&h_c, d_c, sizeof(BASE_TYPE), hipMemcpyDeviceToHost);
	printf("h_c = %f\n", h_c);
}


void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

texture<float, 1, hipReadModeElementType> texRefx;

__global__ void eval_x(float *x, float * h)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < N)
	{
		x[idx] = idx * (*h);
		//printf("x[%i] = %f\n", idx, x[idx]);
	}
}

__global__ void eval_y_rect(float *x, float *res)
{
	__shared__ float temp[THREADS_PER_BLOCK];
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx + 1 <N)
	{
		//printf("temp[%i]-temp[%i] =%f - %f =%f\n", idx + 1, idx, tex1Dfetch(texRefx, (idx + 1)) , tex1Dfetch(texRefx, (idx)),tex1Dfetch(texRefx, (idx + 1))- tex1Dfetch(texRefx, (idx)));
		temp[threadIdx.x] = sqrtf(1 - powf((tex1Dfetch(texRefx, (idx)) + tex1Dfetch(texRefx, (idx + 1))) / 2.0, 2)) *(tex1Dfetch(texRefx, (idx + 1)) - tex1Dfetch(texRefx, (idx)));
		//printf("temp[%i] = %f\n", threadIdx.x, temp[threadIdx.x]);
		__syncthreads();
		if (threadIdx.x == 0)
		{
			float sum = 0.0;
			for (int j = 0; j < blockDim.x; j++)
			{
				//printf("CCCCtemp[%i]-temp[%i] =%f - %f =%f\n", j + 1, j, temp[j + 1], temp[j], temp[j + 1] - temp[j]);
				sum += temp[j];
				//printf("y[] = %f\n",  temp[j]);
			}
			atomicAdd(res, sum);
		}
	}
}

void SecondLab()
{
	int a = 0; int b = 1;
	float h = (b - a) / float(N);
	int memSize = sizeof(float);
	float *d_h;
	hipMalloc((void **)&d_h, memSize);
	hipMemcpy(d_h, &h, memSize, hipMemcpyHostToDevice);

	float *h_x = (float*)malloc(N * memSize);
	for (int i = 0; i < N; i++)
	{
		h_x[i] = a + i*h;
		//printf("h_x[%i] = %f\n",i, h_x[i]);
	}

	float *d_x;
	hipMalloc((void **)&d_x, N * memSize);

	eval_x << <BLOCK_SIZE, THREADS_PER_BLOCK >> > (d_x, d_h);
	hipBindTexture(0, texRefx, d_x, N * memSize);
	checkCUDAError("bind");

	float *d_y;
	hipMalloc((void **)&d_y, memSize);
	float sum2 = 0;
	hipMemcpy(d_y, &sum2, memSize, hipMemcpyHostToDevice);
	//�������� �� �����
	printf("check int on host:\n");
	float *h_y = (float*)malloc(N * memSize);
	float sum = 0;
	for (int j = 0; j < N - 1; j++)
	{
		//printf("fun = %f\n", sqrtf(1 - powf((h_x[j] + h_x[j + 1]) / 2.0, 2)) * (h_x[j + 1] - h_x[j]));
		sum += sqrtf(1 - powf((h_x[j] + h_x[j + 1]) / 2.0, 2)) * (h_x[j + 1] - h_x[j]);
		//printf("h_x[%i] = %f\n",j, h_x[j]);
	}
	printf("int on host = %f:\n", sum * 4);
	eval_y_rect << <BLOCK_SIZE, THREADS_PER_BLOCK >> >(d_x, d_y);
	hipMemcpy(&sum2, d_y, memSize, hipMemcpyDeviceToHost);
	
	//for (int j = 0; j < N; j++)
	//{
	//	sum2 += h_y[j];
	//	//printf("h_y[%i] = %f\n",j, h_y[j]);
	//}
	printf("int on device = %f:\n", sum2 * 4);
	checkCUDAError("hipMemcpy");
	hipUnbindTexture(texRefx);
	checkCUDAError("hipUnbindTexture");
	free(h_x); free(h_y);
	hipFree(d_x); hipFree(d_y); hipFree(d_h);
}


texture<float, 1, hipReadModeElementType> texRefx1;
texture<float, 1, hipReadModeElementType> texRefy1;
texture<float, 1, hipReadModeElementType> texRefx2;
texture<float, 1, hipReadModeElementType> texRefy2;
texture<float, 1, hipReadModeElementType> texRefx3;
texture<float, 1, hipReadModeElementType> texRefy3;
__global__ void scalMult1(BASE_TYPE *C)
{
	__shared__ BASE_TYPE ash[THREADS_PER_BLOCK];
	ash[threadIdx.x] = tex1Dfetch(texRefx1, (blockIdx.x * blockDim.x + threadIdx.x)) * tex1Dfetch(texRefy1, (blockIdx.x * blockDim.x + threadIdx.x));
	__syncthreads();
	if (threadIdx.x == 0)
	{
		BASE_TYPE sum = 0.0;
		for (int j = 0; j < blockDim.x; j++)
		{
			sum += ash[j];
		}
		atomicAdd(C, sum);
	}
}

__global__ void scalMult2(BASE_TYPE *C)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ BASE_TYPE ash[THREADS_PER_BLOCK];
	//printf("tex1Dfetch(texRefx2, (blockIdx.x * blockDim.x + threadIdx.x)) = %f\n", tex1D(texRefx2, float(idx)) );
	//printf("tex1Dfetch(texRefy2, (blockIdx.x * blockDim.x + threadIdx.x)) = %f\n", tex1D(texRefy2, float(idx)) );
	ash[threadIdx.x] = tex1D(texRefx2, float(idx))*  tex1D(texRefy2, float(idx));
	__syncthreads();
	if (threadIdx.x == 0)
	{
		BASE_TYPE sum = 0.0;
		for (int j = 0; j < blockDim.x; j++)
		{
			sum += ash[j];
		}
		atomicAdd(C, sum);
	}
}
__global__ void scalMult3(BASE_TYPE *C)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ BASE_TYPE ash[THREADS_PER_BLOCK];
	//printf("tex1Dfetch(texRefx2, (blockIdx.x * blockDim.x + threadIdx.x)) = %f\n", tex1D(texRefx2, float(idx)) );
	//printf("tex1Dfetch(texRefy2, (blockIdx.x * blockDim.x + threadIdx.x)) = %f\n", tex1D(texRefy2, float(idx)) );
	ash[threadIdx.x] = tex1Dfetch(texRefx3, (blockIdx.x * blockDim.x + threadIdx.x)) *  tex1D(texRefy3, float(idx));
	__syncthreads();
	if (threadIdx.x == 0)
	{
		BASE_TYPE sum = 0.0;
		for (int j = 0; j < blockDim.x; j++)
		{
			sum += ash[j];
		}
		atomicAdd(C, sum);
	}
}

void FirstHome()
{
	//��� ������� ������� � ���. �������
	float *h_res1 = (float*)malloc(sizeof(float));
	float *d_res1;
	hipMalloc((void**)&d_res1, sizeof(float));
	float scRes1 = 0;
	hipMemcpy(d_res1, &scRes1, sizeof(float), hipMemcpyHostToDevice);

	float *d_x1;
	hipMalloc((void**)&d_x1, sizeof(float) * N);
	hipBindTexture(0, texRefx1, d_x1, sizeof(float) * N);

	float *d_y1;
	hipMalloc((void**)&d_y1, sizeof(float) * N);
	hipBindTexture(0, texRefy1, d_y1, sizeof(float) * N);

	float *h_x1 = (float*)malloc(N * sizeof(float));
	float *h_y1 = (float*)malloc(N * sizeof(float));
	for (int i = 0; i != N; i++)
	{
		h_x1[i] = rand() / (BASE_TYPE)RAND_MAX;
		h_y1[i] = rand() / (BASE_TYPE)RAND_MAX;
	}
	//check
	float check_res = 0;
	for (int i = 0; i != N; i++)
	{
		check_res += h_x1[i] * h_y1[i];
	}
	printf("check_res = %f\n", check_res);
	hipMemcpy(d_x1, h_x1, sizeof(float)*N, hipMemcpyHostToDevice);
	hipMemcpy(d_y1, h_y1, sizeof(float)*N, hipMemcpyHostToDevice);
	hipBindTexture(0, texRefx1, d_x1, sizeof(float) * N);
	hipBindTexture(0, texRefy1, d_y1, sizeof(float) * N);
	// �������������� �������
	hipEvent_t start, stop;
	float elapsedTime;
	// ������� �������
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// ������ �������
	hipEventRecord(start, 0);
	// ����� ����
	scalMult1 << <BLOCK_SIZE, THREADS_PER_BLOCK >> > (d_res1);
	hipEventRecord(stop, 0);
	// �������� ���������� ������ ����
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	// ����� ����������
	printf("Time spent executing by the GPU add with 2 lin mem: %.3f millseconds\n", elapsedTime);
	// ����������� �������
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipMemcpy(&scRes1, d_res1, sizeof(float), hipMemcpyDeviceToHost);
	printf("add with 2 lin mem = %f\n", scRes1);
	hipUnbindTexture(texRefx1);
	hipUnbindTexture(texRefy1);

	//��� cudaArr
	hipArray* cuArrayX;
	hipMallocArray(&cuArrayX, &texRefx2.channelDesc, N, 1);
	hipMemcpyToArray(cuArrayX, 0, 0, h_x1, sizeof(float)*N, hipMemcpyHostToDevice);
	hipBindTextureToArray(texRefx2, cuArrayX);
	texRefx2.normalized = false;
	texRefx2.filterMode = hipFilterModePoint;

	hipArray* cuArrayY;
	hipMallocArray(&cuArrayY, &texRefy2.channelDesc, N, 1);
	hipMemcpyToArray(cuArrayY, 0, 0, h_y1, sizeof(float)*N, hipMemcpyHostToDevice);
	hipBindTextureToArray(texRefy2, cuArrayY);
	texRefy2.normalized = false;
	texRefy2.filterMode = hipFilterModePoint;

	// �������� ���������
	scRes1 = 0;
	hipMemcpy(d_res1, &scRes1, sizeof(float), hipMemcpyHostToDevice);

	// �������������� �������
	hipEvent_t start2, stop2;
	float elapsedTime2;
	// ������� �������
	hipEventCreate(&start2);
	hipEventCreate(&stop2);
	// ������ �������
	hipEventRecord(start2, 0);
	// ����� ����
	scalMult2 << <BLOCK_SIZE, THREADS_PER_BLOCK >> > (d_res1);
	hipEventRecord(stop2, 0);
	// �������� ���������� ������ ����
	hipEventSynchronize(stop2);
	hipEventElapsedTime(&elapsedTime2, start2, stop2);
	// ����� ����������
	printf("Time spent executing by the GPU add with 2 cuda arr: %.3f millseconds\n", elapsedTime2);
	// ����������� �������
	hipEventDestroy(start2);
	hipEventDestroy(stop2);
	hipMemcpy(&scRes1, d_res1, sizeof(float), hipMemcpyDeviceToHost);
	
	printf("add with 2 cuda arr = %f\n", scRes1);
	hipUnbindTexture(texRefx2);
	hipUnbindTexture(texRefy2);

	//���������
	hipBindTexture(0, texRefx3, d_x1, sizeof(float) * N);

	hipArray* cuArrayY1;
	hipMallocArray(&cuArrayY1, &texRefy3.channelDesc, N, 1);
	hipMemcpyToArray(cuArrayY1, 0, 0, h_y1, sizeof(float)*N, hipMemcpyHostToDevice);
	hipBindTextureToArray(texRefy3, cuArrayY1);
	texRefy3.normalized = false;
	texRefy3.filterMode = hipFilterModePoint;

	// �������� ���������
	scRes1 = 0;
	hipMemcpy(d_res1, &scRes1, sizeof(float), hipMemcpyHostToDevice);

	// �������������� �������
	hipEvent_t start3, stop3;
	float elapsedTime3;
	// ������� �������
	hipEventCreate(&start3);
	hipEventCreate(&stop3);
	// ������ �������
	hipEventRecord(start3, 0);
	// ����� ����
	scalMult3 << <BLOCK_SIZE, THREADS_PER_BLOCK >> > (d_res1);
	hipEventRecord(stop3, 0);
	// �������� ���������� ������ ����
	hipEventSynchronize(stop3);
	hipEventElapsedTime(&elapsedTime3, start3, stop3);
	// ����� ����������
	printf("Time spent executing by the GPU add with 2 cuda arr: %.3f millseconds\n", elapsedTime3);
	// ����������� �������
	hipEventDestroy(start3);
	hipEventDestroy(stop3);
	hipMemcpy(&scRes1, d_res1, sizeof(float), hipMemcpyDeviceToHost);
	
	printf("add with lin mem and cuda arr = %f\n", scRes1);
	hipUnbindTexture(texRefx3);
	hipUnbindTexture(texRefy3);
}

int main()
{
	//FirstLab();
	//SecondLab();
	FirstHome();
	hipDeviceSynchronize();
	getchar();
	return 0;
}
