#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
using namespace std;


__global__ void Add(int *a, int* b, int* c)
{
	printf("a=%d", *a);
	printf("b=%d", *b);
	*c = (*a) + (*b);
}

int main()
{
	//HelloWorld <<< 2, 5 >>>();
	int a, b, c; // on host
	cin >> a >> b;
	cout << "a = " << a << endl;
	cout << "b = " << b << endl;
	int *devA, *devB, *devC;
	//memory on dev
	hipMalloc((void**)&devA, sizeof(int));
	hipMalloc((void**)&devB, sizeof(int));
	hipMalloc((void**)&devC, sizeof(int));
	//copy host to device
	hipMemcpy(devA, &a, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(devB, &b, sizeof(int), hipMemcpyHostToDevice);
	Add << <1, 1 >> > (devA, devB, devC);
	//copy of the result from device to host
	hipMemcpy(&c, devC, sizeof(int), hipMemcpyDeviceToHost);
	printf("%d + %d = %d\n", a, b, c);
	hipDeviceSynchronize();
	system("Pause");
	return 0;
}