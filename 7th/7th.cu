#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
using namespace std;

#define BLOCK_SIZE 32
// ���, ������� ����� ����� �������� ������
#define BASE_TYPE float
#define THREADS_PER_BLOCK 128
// ������ 
#define N BLOCK_SIZE * THREADS_PER_BLOCK




__global__ void ScalMult(const float *a, const
	float *b, float *c, int numElem)
{
	// ���������� ��� �������� ����� ���������

	// �������� �������� � ����������� ������
	__shared__ float arrShared[THREADS_PER_BLOCK];
	// ����������� �� ���������� ������

	printf("----------------------\n");
	printf("a[%d * %d + %d] = %f\n", blockIdx.x, blockDim.x, threadIdx.x, a[blockIdx.x * blockDim.x + threadIdx.x]);
	printf("b[%d * %d + %d] = %f\n", blockIdx.x, blockDim.x, threadIdx.x, b[blockIdx.x * blockDim.x + threadIdx.x]);
	arrShared[threadIdx.x] = a[blockIdx.x * blockDim.x + threadIdx.x] * b[blockIdx.x * blockDim.x + threadIdx.x];
	// ������������� �����
	__syncthreads();
	// ���������� ���������� ������������
	if (threadIdx.x == 0)
	{
		float sum = 0.0;
		for (int j = 0; j < THREADS_PER_BLOCK; j++)
		{
			sum += arrShared[j];
		}
		/*c[blockIdx.x] = sum;*/
		atomicAdd(c, sum);
	}
}

__global__ void eval_sqrtf(float *x, float *res)
{
	int idx = blockDim.x*blockIdx.x + threadIdx.x;
	if (idx<N)
	{
		*res = sqrtf(*x);
	}

}

__global__ void eval_x(float *x, float * h)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < N)
	{
		x[idx] = idx * (*h);
		//printf("x[%i] = %f\n", idx, x[idx]);
	}
}

__global__ void eval_y_rect(float *x, float *res)
{
	__shared__ float temp[THREADS_PER_BLOCK];
	temp[threadIdx.x] = x[blockIdx.x * blockDim.x + threadIdx.x];
	__syncthreads();
	if (threadIdx.x == 0)
	{
		float sum = 0.0;
		for (int j = 0; j < THREADS_PER_BLOCK - 1; j++)
		{
			sum += sqrtf(1 - powf((temp[j] + temp[j + 1]) / 2.0, 2))*(temp[j + 1] - temp[j]);
			//printf("y[] = %f\n", sqrtf(1 - powf((temp[j] + temp[j + 1]) / 2.0, 2))*(temp[j + 1] - temp[j]) );
		}
		atomicAdd(res, sum);
	}
}

__global__ void eval_y_trapz(float *x, float *res)
{
	__shared__ float temp[THREADS_PER_BLOCK];
	temp[threadIdx.x] = x[blockIdx.x * blockDim.x + threadIdx.x];
	__syncthreads();
	if (threadIdx.x == 0)
	{
		float sum = 0.0;
		for (int j = 0; j < THREADS_PER_BLOCK-1; j++)
		{
			sum += (sqrtf(1 - powf(temp[j], 2)) + sqrtf(1 - powf(temp[j + 1], 2))) / 2.0 * (temp[j + 1] - temp[j]);
			//printf("y[] = %f\n", sqrtf(1 - powf((temp[j] + temp[j + 1]) / 2.0, 2))*(temp[j + 1] - temp[j]));
		}
		atomicAdd(res, sum);
	}
}

__global__ void eval_y_simps(float *x, float *h, float *res)
{
	__shared__ float temp[THREADS_PER_BLOCK];
	temp[threadIdx.x] = x[blockIdx.x * blockDim.x + threadIdx.x];
	__syncthreads();
	if (threadIdx.x == 0)
	{
		float sum = 0.0;
		for (int j = 1; j < THREADS_PER_BLOCK - 1; j = j + 2)
		{
			sum += (*h / 3.0 * (sqrtf(1 - powf(temp[j - 1], 2)) + 4.0*sqrtf(1 - powf(temp[j], 2)) + sqrtf(1 - powf(temp[j - 1], 2))));
			//printf("y[] = %f\n", sqrtf(1 - powf((temp[j] + temp[j + 1]) / 2.0, 2))*(temp[j + 1] - temp[j]));
		}
		atomicAdd(res, sum);
	}
}


void FirstLab()
{
	// ���������� �� �����
	float *h_a;
	h_a = (float*)malloc(N * sizeof(float));
	float *h_b;
	h_b = (float *)malloc(N * sizeof(float));
	float h_c = 0;
	for (int i = 0; i< N; i++)
	{
		h_a[i] = rand() % 10 + 1;
		printf("a[%d] = %f\n", i, h_a[i]);
		h_b[i] = rand() % 10 + 1;
		printf("b[%d] = %f\n", i, h_b[i]);
	}
	printf("Arrays' size = %i\n", N);
	for (int i = 0; i != N; i++)
	{
		float temp = h_a[i] * h_b[i];
		h_c += temp;
	}

	printf("host result = %f\n", h_c);

	h_c = 0;
	// �� �������
	float * d_a;
	hipMalloc((void**)&d_a, N * sizeof(float));
	float * d_b;
	hipMalloc((void**)&d_b, N * sizeof(float));
	float *d_c;
	hipMalloc((void**)&d_c, sizeof(float));
	hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_c, &h_c, sizeof(float), hipMemcpyHostToDevice);
	ScalMult << <BLOCK_SIZE, THREADS_PER_BLOCK >> > (d_a, d_b, d_c, N);
	//float *h_cc = (float*)malloc(sizeof(float));
	//hipMemcpy(h_cc, &d_c,  sizeof(float), hipMemcpyDeviceToHost);
	float res = 0;

	hipMemcpy(&res, d_c, sizeof(float), hipMemcpyDeviceToHost);
	/*for (int i = 0; i != N; i++)
	{
		res += h_cc[i];
	}*/
	printf("device result= %f\n", res);


}

void FirstHome()
{
	int a = 0, b = 1;
	int *d_a, *d_b;
	hipMalloc((void**)&d_a, sizeof(int));
	hipMalloc((void**)&d_b, sizeof(int));
	hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

	float *x, *y;
	float *d_x, *d_y;
	x = (float*)malloc(sizeof(float)*N);
	y = (float*)malloc(sizeof(float)*N);
	hipMalloc((void**)&d_x, sizeof(float)*N);
	hipMalloc((void**)&d_y, sizeof(float)*N);

	float res1 = 0;
	float * d_res1;
	hipMalloc((void**)&d_res1, sizeof(float));
	hipMemcpy(d_res1, &res1, sizeof(float), hipMemcpyHostToDevice);
	float res2 = 0;
	float * d_res2;
	hipMalloc((void**)&d_res2, sizeof(float));
	hipMemcpy(d_res2, &res2, sizeof(float), hipMemcpyHostToDevice);
	float res3 = 0;
	float * d_res3;
	hipMalloc((void**)&d_res3, sizeof(float));
	hipMemcpy(d_res3, &res3, sizeof(float), hipMemcpyHostToDevice);
	float h = (b - a)*1.0 / (N);
	printf("h = %f\n", h);
	float * d_h;
	hipMalloc((void**)&d_h, sizeof(float));
	hipMemcpy(d_h, &h, sizeof(float), hipMemcpyHostToDevice);

	eval_x << <BLOCK_SIZE, THREADS_PER_BLOCK >> > (d_x, d_h);
	eval_y_rect << <BLOCK_SIZE, THREADS_PER_BLOCK >> > (d_x, d_res1);
	hipMemcpy(&res1, d_res1, sizeof(float), hipMemcpyDeviceToHost);
	eval_y_trapz << <BLOCK_SIZE, THREADS_PER_BLOCK >> > (d_x, d_res2);
	hipMemcpy(&res2, d_res2, sizeof(float), hipMemcpyDeviceToHost);
	eval_y_simps << <BLOCK_SIZE, THREADS_PER_BLOCK >> > (d_x, d_h, d_res3);
	hipMemcpy(&res3, d_res3, sizeof(float), hipMemcpyDeviceToHost);
	res1 *= 4.0;
	res2 *= 4.0;
	res3 *= 4.0;
	printf("res rect = %f\n", res1);
	printf("res trapz = %f\n", res2);
	printf("res simpson = %f\n", res3);
	free(x); free(y);
	hipFree(d_a); hipFree(d_b); hipFree(d_h); hipFree(d_res1); hipFree(d_res2); hipFree(d_res3);

	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
		printf("%s", hipGetErrorString(err));
}

void SecondHome()
{
	float *x;
	float *d_x;
	x = (float*)malloc(sizeof(float)*N);
	for (int i = 0; i < N; ++i) {
		x[i] = rand() / (float)RAND_MAX;
		//printf("x[%i] = %f\n", i, x[i]);
	}
	hipMalloc((void**)&d_x, sizeof(float)*N);
	hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);

	float *h_res1 = (float*)malloc(sizeof(float));
	float * d_res1; float * dd_res1;
	hipMalloc((void**)&d_res1, N * sizeof(float));
	hipMalloc((void**)&dd_res1, sizeof(float));
	ScalMult << <BLOCK_SIZE, THREADS_PER_BLOCK >> > (d_x, d_x, d_res1, N);
	eval_sqrtf << <1, 1 >> > (d_res1, dd_res1);
	hipMemcpy(h_res1, dd_res1, sizeof(float), hipMemcpyDeviceToHost);

	float sum = 0;
	for (int i = 0; i < N; ++i) {
		sum += x[i] * x[i];
	}

	sum = sqrtf(sum);
	printf("sum  = %f\n", sum);
	//res1 = powf(res1,0.5);

	printf("res  = %f\n", *h_res1);
	free(x);
	hipFree(d_res1);

	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
		printf("%s", hipGetErrorString(err));
}

int main()
{
	//FirstLab();
	//FirstHome();
	SecondHome();
	hipDeviceSynchronize();
	getchar();
	return 0;
}
